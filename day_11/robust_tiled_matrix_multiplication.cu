#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16


__global__ void matmul_tiled(
    const float* __restrict__ A, const float* __restrict__ B, float* C,
    int M, int K, int N
)
{
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float acc = 0.0f;
    for(int t = 0; t<(K + TILE_SIZE - 1)/TILE_SIZE; t++){
        
        int tiled_col = t*TILE_SIZE + threadIdx.x;
        int tiled_row = t*TILE_SIZE + threadIdx.y;

        As[threadIdx.y][threadIdx.x] = 
            (row < M && tiled_col < K) ? A[row * K + tiled_col] : 0.0f;
        
        Bs[threadIdx.y][threadIdx.x] = 
            (col < N && tiled_row < K) ? B[tiled_row * N + col] : 0.0f;
        __syncthreads();

        for(int k = 0; k<TILE_SIZE; k++){
            acc+=As[threadIdx.y][k]*Bs[k][threadIdx.x];
        }
        __syncthreads();
    }
    
    if(row<M && col<N)
        C[row*N + col] = acc;
}

void initialize(float* mat, int rows, int col){
    for(int i = 0; i<rows*col; i++){
        mat[i] = static_cast<float>(rand()%5 + 1);
    }
}

int main(){
    int M = 50, K = 78, N = 63;

    size_t size_B = M * K * sizeof(float);
    size_t size_A = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    float* h_A  = (float*)malloc(size_A);
    float* h_B  = (float*)malloc(size_B);
    float* h_C  = (float*)malloc(size_C);

    initialize(h_A, M, K);
    initialize(h_B, K, N);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    dim3 threads(TILE_SIZE, TILE_SIZE);
    // C is of shape M x N, so M rows and N cols, that is, M is on y axis
    // and N is on the x axis 
    dim3 blocks((N + TILE_SIZE -1)/TILE_SIZE, (M + TILE_SIZE -1)/TILE_SIZE);

    matmul_tiled<<<blocks, threads>>>(d_A, d_B, d_C, M, K, N);
