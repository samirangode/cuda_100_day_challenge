
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

#define N 4
#define D 4
#define TILE 2

__global__ void conv2d_kernel(const float* img,
                              const float* kernel,
                              float* output,
                              int H, int W, int K
){
    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;
    int outH = H - K + 1, outW = W - K + 1;

    if(out_x < outW && out_y < outH){
        float sum = 0.0f;
        for(int i = 0; i<K; i++){
            for(int j = 0; j<K; j++){
                int img_x = out_x + j;
                int img_y = out_y + i;
                sum += img[img_y * W + img_x] * kernel[i*K + j];
            }
        }
        output[out_y * outW + out_x] = sum;
    }

}


int main(){
    int H = 64, W = 64, K = 4;
    int outH = H-K+1, outW = W-K+1;

    float img[H*W];
    float kernel[K*K];
    float output[outH*outW];

    float val = 1.0;
    for(int i = 0; i<H; i++){
        for(int j = 0; j<W; j++){
            img[i*W + j] = val;
            val+=1;
        }
    }
    val = 1.0;
    for(int i = 0; i<K; i++){
        for(int j = 0; j<K; j++){
            kernel[i*K + j] = val;
            val+=1;
        }
    }

    float* img_device, * kernel_device, *output_device;
    hipMalloc(&img_device, H * W * sizeof(float));
    hipMalloc(&kernel_device, K * K * sizeof(float));
    hipMalloc(&output_device, outH * outW * sizeof(float));

    // transfer
    hipMemcpy(img_device, img, H * W * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(kernel_device, kernel, K * K * sizeof(float), hipMemcpyHostToDevice);


    // Launching CUDA Kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((outW + blockDim.x - 1)/blockDim.x,
                 (outH + blockDim.y - 1)/blockDim.y);
    conv2d_kernel<<<gridDim, blockDim>>>(img_device, kernel_device, output_device, H, W, K);

    hipMemcpy(output, output_device, outH * outW * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i<outH; i++){
        for(int j = 0; j<outW; j++){
            std::cout<<output[i*outW + j]<<" ";
        }
        std::cout<<std::endl;
    }

    hipFree(img_device); hipFree(kernel_device); hipFree(output_device);

    return 0;
}
