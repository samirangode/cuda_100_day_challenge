#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>


#define TILE_SIZE 16


__global__ void matmul_tiled(
    const float* __restrict__ A, const float* __restrict__ B, float* C,
    int M, int K, int N
)
{
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float acc = 0.0f;
    for(int t = 0; t<(K + TILE_SIZE - 1)/TILE_SIZE; t++){
        
        int tiled_col = t*TILE_SIZE + threadIdx.x;
        int tiled_row = t*TILE_SIZE + threadIdx.y;

        As[threadIdx.y][threadIdx.x] = 
            (row < M && tiled_col < K) ? A[row * K + tiled_col] : 0.0f;
        
        Bs[threadIdx.y][threadIdx.x] = 
            (col < N && tiled_row < K) ? B[tiled_row * N + col] : 0.0f;
        __syncthreads();

        for(int k = 0; k<TILE_SIZE; k++){
            acc+=As[threadIdx.y][k]*Bs[k][threadIdx.x];
        }
        __syncthreads();
    }
    
    if(row<M && col<N)
        C[row*N + col] = acc;
}

void initialize(float* mat, int rows, int col){
    for(int i = 0; i<rows*col; i++){
        mat[i] = static_cast<float>(rand()%5 + 1);
    }
}

void matmul_cpu(const float* A, const float* B, float* C, int M, int K, int N){
    for(int i = 0; i< M; i++){
        for(int j = 0; j<N; j++){
            float acc = 0.0f;
            for(int k = 0; k< K; k++){
                acc += A[i*K + k] * B[k*N + j];
            }
            C[i*N + j] = acc;
        }
    }
}


int main(){
    int M = 50, K = 78, N = 63;

    size_t size_B = M * K * sizeof(float);
    size_t size_A = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    float* h_A  = (float*)malloc(size_A);
    float* h_B  = (float*)malloc(size_B);
    float* h_C  = (float*)malloc(size_C);
    float* h_C_custom = (float*)malloc(size_C);
    float* h_C_cpu  = (float*)malloc(size_C);

    initialize(h_A, M, K);
    initialize(h_B, K, N);

    // Declare cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate();

    // Allocate memory on GPU
    float *d_A, *d_B, *d_C, *d_C_custom;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    hipMalloc(&d_C_custom, size_C_custom);
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // For cuBLAS, alpha and beta (scalars) can be passed as pointers
    const float alpha = 1.0f;
    const float beta = 0.0f;
    // Benchmark cuBLAS
    auto start = std::chrono::high_resolution_clock::now();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    double cuBLAS_time = std::chrono::duration<double, std::milli>(end - start).count();
    std::cout << "cuBLAS matmul time: " << cuBLAS_time << " ms" << std::endl;

    // Benchmark custom kernel
    auto start_gpu = std::chrono::high_resolution_clock::now();
    matmul_tiled<<<blocks, threads>>>(d_A, d_B, d_C_custom, M, K, N);
    hipDeviceSynchronize();
    auto end_gpu = std::chrono::high_resolution_clock::now();
    double gpu_time = std::chrono::duration<double, std::milli>(end_gpu - start_gpu).count();
    std::cout << "GPU matmul time (your kernel): " << gpu_time << " ms" << std::endl;
    hipMemcpy(h_C, d_C_custom, size_C, hipMemcpyDeviceToHost);


    matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);


    double max_abs_error = 0.0;
    for (int i = 0; i < M * N; ++i) {
        double err = std::abs(h_C[i] - h_C_cpu[i]);  // h_C_cpu is the CPU result
        if (err > max_abs_error) max_abs_error = err;
    }
    std::cout << "Max absolute error: " << max_abs_error << std::endl;

    hipFree(d_A); hipFree(d_B); hipFree(d_C); hipFree(d_C_custom);
    free(h_A); free(h_B); free(h_C); free(h_C_custom); free(h_C_cpu);
    hipblasDestroy(handle);
    return 0;

}